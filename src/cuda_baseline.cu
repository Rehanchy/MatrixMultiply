#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>
#include <ctime>
#include <immintrin.h>
using namespace std;

int N = 0;
inline void gemm_verify(float *C_cpu, float *C_gpu); // you can use inline function
float X = 11.4514;
__global__ void gemm_block(float* A, float* B, float* C, int N);
inline void gemm_Blas(float *A, float *B, float *C);
 
// <2d grid, 1d block>
#define get_tid() ((blockIA_gpu.y*gridDim.x + blockIA_gpu.x)*blockDim.x + threadIA_gpu.x)
#define get_bid() (blockIA_gpu.y*gridDim.x + blockIA_gpu.x)
 
int main(int argc, char* argv[])
{
	int num = 0;
    int n_block = 0, n_grid = 0;
    float* A_gpu = NULL, *A_cpu = NULL;
	float* B_gpu = NULL, *B_cpu = NULL;
	float* C_gpu = NULL, *C_cpu = NULL, *C_cpu2 = NULL;
    srand(time(0));
    num = atoi(argv[1]);
    cout << "Specified N as " << num << endl;
    N = (1 << num);
    n_block = atoi(argv[2]);
    cout << "Specified threads each block as " << n_block << ", " << n_block << endl;
    n_grid = int(N / n_block);
    cout << "Thus block number is " << n_grid<< ", " << n_grid<< endl;
    // initialize A, B, C
    A_cpu = new float[N * N];
    B_cpu = new float[N * N];
    C_cpu = new float[N * N];
    C_cpu2 = new float[N * N];
    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < N; j++)
        {
            A_cpu[i * N + j] = (float)rand() / (float)(RAND_MAX) * X;
            B_cpu[i * N + j] = (float)rand() / (float)(RAND_MAX) * X;
        }
    }
	int nbytes = N * N * sizeof(float);
	dim3 dimGrid(n_grid, n_grid); // how many thread blocks
	dim3 dimBlock(n_block, n_block); // how many threads in each block
	hipError_t cudaStatus = hipSetDevice(0); // one GPU
 
 
	/* allocate gpu memory */
	hipMalloc((void**)&A_gpu, nbytes);
 
	hipMalloc((void**)&B_gpu, nbytes);
 
	hipMalloc((void**)&C_gpu, nbytes);

	/* copy data to gpu*/
	hipMemcpy(A_gpu, A_cpu, nbytes, hipMemcpyHostToDevice);
 
	hipMemcpy(B_gpu, B_cpu, nbytes, hipMemcpyHostToDevice);

 
	// call for gpu
	hipDeviceSynchronize();
	gemm_block <<< dimGrid, dimBlock >>> (A_gpu, B_gpu, C_gpu, N);
 
	hipDeviceSynchronize();
 
	// call for cpu
    hipMemcpy(C_cpu2, C_gpu, nbytes, hipMemcpyDeviceToHost);
    if(N <= 10)
    {
        cout << "Small Matrix, apply verification" << endl;
        gemm_Blas(A_cpu, B_cpu, C_cpu);
        gemm_verify(C_cpu, C_cpu2);
        cout << "Verify Success" << endl;
    }
    else
    {
        cout << "Large Matrix, abort verification" << endl;
    }
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	delete [] A_cpu;
	delete [] B_cpu;
	delete [] C_cpu;
    delete [] C_cpu2;
	return 0;
}

inline void gemm_verify(float* C_cpu, float* C_gpu)
{
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
            if(abs(C_gpu[i * N + j]- C_cpu[i * N + j])>1e-5)
            {
                printf("%.12f, %.12f", C_gpu[i * N + j], C_cpu[i * N + j]);
                cout << "error" << endl;
                exit(0);
            }
		}
	}
}

// device code
__global__ void gemm_block(float* A, float* B, float* C, int N)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; // thread index COL
	int y = blockIdx.y * blockDim.y + threadIdx.y; // thread index ROW
 
	float sum = 0, a, b;
	for (int i = 0; i < N; ++i)
	{
		a = A[y * N + i];
		b = B[i * N + x];
		sum += a * b;
	}
	C[y * N + x] = sum;
}

void addDot8x8Pack(float *A, float *B, float *C) 
{
    int p;
	float *Packed_ptr = B;
	__m256 Vec1 = _mm256_setzero_ps();
	__m256 Vec2 = _mm256_setzero_ps();
	__m256 Vec3 = _mm256_setzero_ps();
	__m256 Vec4 = _mm256_setzero_ps();
	__m256 Vec5 = _mm256_setzero_ps();
	__m256 Vec6 = _mm256_setzero_ps();
	__m256 Vec7 = _mm256_setzero_ps();
	__m256 Vec8 = _mm256_setzero_ps();
    
	for (p = 0; p < N; p++) 
    {
		__m256 a0 = _mm256_set1_ps(*(A + p));
		__m256 a1 = _mm256_set1_ps(*(A + N + p));
		__m256 a2 = _mm256_set1_ps(*(A + 2 * N + p));
		__m256 a3 = _mm256_set1_ps(*(A + 3 * N + p));
		__m256 a4 = _mm256_set1_ps(*(A + 4 * N + p));
		__m256 a5 = _mm256_set1_ps(*(A + 5 * N + p));
		__m256 a6 = _mm256_set1_ps(*(A + 6 * N + p));
		__m256 a7 = _mm256_set1_ps(*(A + 7 * N + p)); // each time get one col of A(8 float)

		__m256 bp = _mm256_loadu_ps(Packed_ptr); // load 8 float from B(packed)
                                                // calculate
		Vec1 = _mm256_fmadd_ps(a0, bp, Vec1);
		Vec2 = _mm256_fmadd_ps(a1, bp, Vec2);
		Vec3 = _mm256_fmadd_ps(a2, bp, Vec3);
		Vec4 = _mm256_fmadd_ps(a3, bp, Vec4);
		Vec5 = _mm256_fmadd_ps(a4, bp, Vec5);
		Vec6 = _mm256_fmadd_ps(a5, bp, Vec6);
		Vec7 = _mm256_fmadd_ps(a6, bp, Vec7);
		Vec8 = _mm256_fmadd_ps(a7, bp, Vec8);

		Packed_ptr += 8;
	}
    // 8rows of A * 8cols of B
	_mm256_storeu_ps(C, Vec1);
	_mm256_storeu_ps(C + N, Vec2);
	_mm256_storeu_ps(C + 2 * N, Vec3);
	_mm256_storeu_ps(C + 3 * N, Vec4);
	_mm256_storeu_ps(C + 4 * N, Vec5);
	_mm256_storeu_ps(C + 5 * N, Vec6);
	_mm256_storeu_ps(C + 6 * N, Vec7);
	_mm256_storeu_ps(C + 7 * N, Vec8);
}

void PackedMatrix(int j, float *input, float *output) 
{
	for (int i = 0; i < N; i++) // get 8 cols of B each time, improve cache performance
		memcpy(output + i * 8, input + i * N + j, sizeof(float) * 8); // MAKE IT A LONG LINE
}

// impressed by openBLAS
inline void gemm_Blas(float *A, float *B, float *C)
{
    float *PackedB = new float[8*N];
    for(int j = 0; j < N; j+=8)
    {
        PackedMatrix(j, B, PackedB);
        
        for(int i = 0; i < N; i+=8)
        {
            addDot8x8Pack(A + i * N, PackedB, C + i * N + j);
        }
    }
}